#include "hip/hip_runtime.h"
#include "matsave.h"
#include "nr3plus.h"
using namespace std;
#define H2D hipMemcpyHostToDevice
#define D2H hipMemcpyDeviceToHost
#define DIM 1000

__device__
Int julia(Int x, Int y, float scale) {
    Int i;
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    float jy = scale * (float)(DIM/2 - y)/(DIM/2);
    Complex c(-0.8, 0.15745);
    Complex a(jx, jy);

    for (i=0; i<200; i++) {
        a = a * a + c;
        if (abs(a) > 31.62)
            return 0;
    }
    return 1;
}

__global__
void kernel(Uchar *ptr, float scale) {
    Int indxy, ind = blockIdx.x*blockDim.x + threadIdx.x;
    Int stride = blockDim.x*gridDim.x;
    Int x,y;
    for(indxy = ind; indxy < DIM*DIM; indxy += stride){
        x = indxy%DIM; y = indxy/DIM;
        Int juliaValue = julia( x, y, scale);
        ptr[x + y * DIM] = (Uchar)(255*juliaValue);
    }
}

int main( void ) {
    int i, img_size = DIM*DIM*sizeof(Uchar);
    float scale = 1.5;
    string str;
    MatUchar bitmap(DIM, DIM);
    Uchar *dev_bitmap;

    hipMalloc( (void**)&dev_bitmap, img_size );

    for (i = 0; i < 150; ++i){
        scale *= 0.95;
        kernel<<<320,32>>>( dev_bitmap, scale );
        hipMemcpy(bitmap[0], dev_bitmap, img_size, D2H);
        str = to_string(i);
        MATFile *pfile = matOpen((str + ".julia.mat").c_str(), "w");
        matsave(bitmap, "julia", pfile);
        matClose(pfile);
    }
    
    hipFree( dev_bitmap );
}
