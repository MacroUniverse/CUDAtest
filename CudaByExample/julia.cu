#include "hip/hip_runtime.h"
#include "matsave.h"
#include "nr3plus.h"
using namespace std;

// to write mat file
/* 
MATFile *pfile = matOpen("julia.mat", "w");
matsave(x, "x", pfile);
matClose(pfile);
*/

#define DIM 4000

__device__
int julia( int x, int y ) {
    int i;
    const float scale = 1.5;
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    float jy = scale * (float)(DIM/2 - y)/(DIM/2);
    Complex c(-0.8, 0.156);
    Complex a(jx, jy);

    for (i=0; i<200; i++) {
        a = a * a + c;
        if (abs(a) > 31.62)
            return 0;
    }
    return 1;
}

__global__
void kernel( Uchar *ptr ) {
    Int x = blockIdx.x;
    Int y = blockIdx.y;
    Int juliaValue = julia( x, y );
    ptr[x + y * gridDim.x] = (Uchar)(255*juliaValue);
}

int main( void ) {
    int img_size = DIM*DIM*sizeof(Uchar);
    MatUchar bitmap(DIM, DIM);
    Uchar *dev_bitmap;

    hipMalloc( (void**)&dev_bitmap, img_size );

    dim3 grid(DIM,DIM);
    kernel<<<grid,1>>>( dev_bitmap );

    hipMemcpy( bitmap[0], dev_bitmap, img_size,
                        hipMemcpyDeviceToHost );
    
    MATFile *pfile = matOpen("julia.mat", "w");
    matsave(bitmap, "julia", pfile);
    matClose(pfile);
    hipFree( dev_bitmap );
}

