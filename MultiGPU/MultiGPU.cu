#include "hip/hip_runtime.h"
#ifdef _MSC_VER
#include "hip/hip_runtime.h"
#include ""
#endif
#include <iostream>

using namespace std;

__global__
void add1(int *x, int thread)
{
	x[thread] += 1;
}

int main()
{
	int i, N = 3;
	int *x, *dev_x;
	hipSetDevice(0);
	hipSetDeviceFlags(hipDeviceMapHost);
	hipHostAlloc((void**)&x, N*sizeof(int),
		hipHostMallocWriteCombined |
		hipHostMallocPortable |
		hipHostMallocMapped);
	for (i = 0; i < N; ++i) {
		x[i] = i;
	}
	hipHostGetDevicePointer(&dev_x, x, 0);
	
	#pragma omp parallel for
	for (i = 0; i < N; ++i) {
		if (i != 0) {
			hipSetDevice(i);
			hipSetDeviceFlags(hipDeviceMapHost);
		}
		add1<<<1,1>>>(dev_x, i);
		hipDeviceSynchronize();
	}

	cout << "the result is: " << x[0] << " " << x[1] << " " << x[2] << endl;
}