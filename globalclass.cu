#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
using namespace std;

struct data
{
	int i;
};

__device__ data devdata;

__device__ double xmax = 3.14;
double xmax_;

__global__
void fun()
{
	printf(" xmax = %f\n", xmax);
	devdata.i = 314;
	printf(" devdata.i = %d\n", devdata.i);
}

int main()
{
	fun<<<1,1>>>();
	hipDeviceSynchronize();
	xmax_ = 5.;
	cout << "xmax_ = " << xmax_ << endl;
	data mydata;
	mydata.i = 100;
	cout << "mydata.i = " << mydata.i << endl;
}
