#include "hip/hip_runtime.h"
#include "nr3.h"
#include "nr3plus.h"
using namespace std;

// set pointers in v[i][j] for 3D matrix
__global__
void setMat3DComplex(Complex ***v, Complex **v_0, Complex *v_00, const Int Ni, const Int Nj, const Int Nk)
{
	Int i,j;
	v[0] = v_0; v[0][0] = v_00;
	for(j=1; j<Nj; ++j) v[0][j] = v[0][j-1] + Nk;
	for(i=1; i<Ni; ++i) {
		v[i] = v[i-1] + Nj;
		v[i][0] = v[i-1][0] + Nj*Nk;
		for(j=1; j<Nj; ++j) v[i][j] = v[i][j-1] + Nk;
	}
}


// allocate a 3D matrix in GPU
// every 3D matrix need to have 3 pointers in the host
void cudaNewMat3DComplex(Complex ***&v, Complex **&v_0, Complex *&v_00, const Int Ni, const Int Nj, const Int Nk)
{
	hipMalloc((void****)&v, Ni*sizeof(Complex**));
	hipMalloc((void***)&v_0, Ni*Nj*sizeof(Complex*));
	hipMalloc((void**)&v_00, Ni*Nj*Nk*sizeof(Complex));
	setMat3DComplex<<<1,1>>>(v, v_0, v_00, Ni, Nj, Nk);
}

// deallocate a 3D matrix
void cudaDeleteMat3DComplex(Complex ***&v, Complex **&v_0, Complex *&v_00)
{
	hipFree(v); hipFree(v_0); hipFree(v_00);
}

// propagate the wave function in z direction
__global__
void cn1Dz(Complex ***psi, const Int Nx, const Int Ny, const Int Nz)
{
	Int i,j,k;
	for(i=0; i<Nx; ++i)
	for(j=0; j<Ny; ++j)
	for(k=0; k<Nz; ++k)
		psi[i][j][k] += Complex(1.1, 1.1);
}

int main()
{
	Int i, j, k, size, Nx = 100, Ny = 100, Nz = 100;
	Doub err{0.};
	Mat3DComplex psi(Nx,Ny,Nz);
	Complex ***psi_d, **psi_d_0, *psi_d_00; 
	cudaNewMat3DComplex(psi_d, psi_d_0, psi_d_00, Nx, Ny, Nz);

	size = Nx*Ny*Nz*sizeof(Complex);

	for(i=0;i<Nx;++i)
	for(j=0;j<Ny;++j)
	for(k=0;k<Nz;++k)
		psi[i][j][k] = Complex(0., 0.);

	hipMemcpy(psi_d_00, psi[0][0], size, hipMemcpyHostToDevice);
	
	cn1Dz<<<1,1>>>(psi_d, Nx, Ny, Nz);

	hipMemcpy(psi[0][0], psi_d_00, size, hipMemcpyDeviceToHost);

	for(i=0;i<Nx;++i)
	for(j=0;j<Ny;++j)
	for(k=0;k<Nz;++k)
		err += abs(psi[i][j][k] - Complex(1.1,1.1));

	cout << "err =  " << err << endl;

	cudaDeleteMat3DComplex(psi_d, psi_d_0, psi_d_00);
}
