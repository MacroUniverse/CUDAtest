#include "hip/hip_runtime.h"
#include "nr3plus.h"
using namespace std;

__device__ Int Nx, Nxprint, Ny, Nyprint, Nz, Nzprint, Nt, Ntprint,
	NFadeX1, NFadeX2, NFadeY1, NFadeY2, NFadeZ1, NFadeZ2, NDetecX1,
	NDetecX2, NDetecY1, NDetecY2, NDetecZ1, NDetecZ2, Npx, Npy, Npz,
	NE;
__device__ Doub xmin, xmax, ymin, ymax, zmin, zmax, tmin, tmax, dx,
	dy, dz, dt, fadeX1, fadeX2, fadeY1, fadeY2, fadeZ1, fadeZ2, xc,
	yc, zc, Asoft, asoft, E0x, sigmatx, omegax, lambdax, tcx, E0y,
	sigmaty, omegay, lambday, tcy, pxmin, pxmax, pymin, pymax, Emin,
	Emax;
__device__ Doub *x, *y, *z, *t;

struct Cn3Dparam
{
	Int Nx, Nxprint, Ny, Nyprint, Nz, Nzprint, Nt, Ntprint,
		NFadeX1, NFadeX2, NFadeY1, NFadeY2, NFadeZ1, NFadeZ2, NDetecX1,
		NDetecX2, NDetecY1, NDetecY2, NDetecZ1, NDetecZ2, Npx, Npy, Npz,
		NE;
	Doub xmin, xmax, ymin, ymax, zmin, zmax, tmin, tmax, dx,
		dy, dz, dt, fadeX1, fadeX2, fadeY1, fadeY2, fadeZ1, fadeZ2, xc,
		yc, zc, Asoft, asoft, E0x, sigmatx, omegax, lambdax, tcx, E0y,
		sigmaty, omegay, lambday, tcy, pxmin, pxmax, pymin, pymax, Emin,
		Emax;
	VecDoub x, y, z, t;
};

Cn3Dparam h;


// set pointers in v[i][j] for 3D matrix
__global__
void setMat3DComplex(Complex ***v, Complex **v_0, Complex *v_00, const Int Ni, const Int Nj, const Int Nk)
{
	Int ind = blockIdx.x*blockDim.x + threadIdx.x;
	if (ind < Ni)
		v[ind] = v_0 + Nj*ind;
	else if (ind < Ni*(Nj+1)) {
		Int j = ind - Ni;
		v_0[j] = v_00 + Nk*j;
	}
}

// allocate a 3D matrix in GPU
// every 3D matrix need to have 3 pointers in the host
void cudaNewMat3DComplex(Complex ***&v, Complex **&v_0, Complex *&v_00, const Int Ni, const Int Nj, const Int Nk)
{
	hipMalloc((void****)&v, Ni*sizeof(Complex**));
	hipMalloc((void***)&v_0, Ni*Nj*sizeof(Complex*));
	hipMalloc((void**)&v_00, Ni*Nj*Nk*sizeof(Complex));
	setMat3DComplex<<<(Ni*(Nj+1)+255)/256,256>>>(v, v_0, v_00, Ni, Nj, Nk);
}

// deallocate a 3D matrix
void cudaDeleteMat3DComplex(Complex ***&v, Complex **&v_0, Complex *&v_00)
{
	hipFree(v); hipFree(v_0); hipFree(v_00);
}

__global__
void devInitialize(Cn3Dparam h, Doub *x_d, Doub *y_d, Doub *z_d)
{
	xmin = h.xmin; xmax = h.xmax; Nx = h.Nx;
	ymin = h.ymin; ymax = h.ymax; Ny = h.Ny;
	zmin = h.zmin; zmax = h.zmax; Nz = h.Nz;
	tmin = h.tmin; tmax = h.tmax; Nt = h.Nt;

	x = x_d; y = y_d; z = z_d;
}

void Initialize(Mat3DComplex_O &psi, Complex ***&psi_d, Complex **&psi_d_0,
	Complex *&psi_d_00, Doub *&x_d, Doub *&y_d, Doub *&z_d)
{
	Int i, j, k;

	h.xmin = -5.; h.xmax = 5.; h.Nx = 11;
	h.ymin = -5.; h.ymax = 5.; h.Ny = 11;
	h.zmin = -5.; h.zmax = 5.; h.Nz = 11;
	h.tmin =  0.; h.tmax = 1.; h.Nt = 11;

	linspace(h.x,h.xmin,h.xmax,h.Nx);
	linspace(h.y,h.ymin,h.ymax,h.Ny);
	linspace(h.z,h.zmin,h.zmax,h.Nz);
	linspace(h.t,h.tmin,h.tmax,h.Nt);

	hipMalloc((void**)&x_d, h.Nx*sizeof(Doub));
	hipMalloc((void**)&y_d, h.Ny*sizeof(Doub));
	hipMalloc((void**)&z_d, h.Nz*sizeof(Doub));
	hipMemcpy(x_d, &h.x[0], h.Nx*sizeof(Doub), hipMemcpyHostToDevice);
	hipMemcpy(y_d, &h.y[0], h.Ny*sizeof(Doub), hipMemcpyHostToDevice);
	hipMemcpy(z_d, &h.z[0], h.Nz*sizeof(Doub), hipMemcpyHostToDevice);

	cudaNewMat3DComplex(psi_d, psi_d_0, psi_d_00, h.Nx, h.Ny, h.Nz);

	psi.resize(h.Nx, h.Ny, h.Nz);

	for(i=0;i<h.Nx;++i)
	for(j=0;j<h.Ny;++j)
	for(k=0;k<h.Nz;++k)
		psi[i][j][k] = Complex(0., 0.);

	devInitialize<<<1,1>>>(h, x_d, y_d, z_d);

	
	hipMemcpy(psi_d_00, psi[0][0], h.Nx*h.Ny*h.Nz*sizeof(Complex), hipMemcpyHostToDevice);
}

// propagate the wave function in z direction
__global__
void cn1Dz(Complex ***psi)
{
	Int i,j,k;
	Doub temp;
	for(i=0; i<Nx; ++i)
	for(j=0; j<Ny; ++j)
	for(k=0; k<Nz; ++k) {
		temp = Ny*Nz*i + Nz*j + k;
		psi[i][j][k] += Complex(temp, temp);
	}
}

int main()
{
	Int i, j, k, size;
	Doub err{0.}, temp;
	Doub *x_d, *y_d, *z_d; // corresponds to h.x, h.y, h.z
	Mat3DComplex psi;
	Complex ***psi_d, **psi_d_0, *psi_d_00;
	cout << "in main()" << endl;

	Initialize(psi, psi_d, psi_d_0, psi_d_00, x_d, y_d, z_d);

	size = h.Nx*h.Ny*h.Nz*sizeof(Complex);

	
	//hipDeviceSynchronize();

	cn1Dz<<<1,1>>>(psi_d);

	hipMemcpy(psi[0][0], psi_d_00, size, hipMemcpyDeviceToHost);

	for(i=0;i<h.Nx;++i)
	for(j=0;j<h.Ny;++j)
	for(k=0;k<h.Nz;++k) {
		temp = h.Ny*h.Nz*i + h.Nz*j + k;
		err += abs(psi[i][j][k] - Complex(temp,temp));
	}

	cout << "err =  " << err << endl;

	cudaDeleteMat3DComplex(psi_d, psi_d_0, psi_d_00);
}
