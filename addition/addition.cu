#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
#define N 512
using namespace std;


__global__ void add(double *a, double *b, double *c)
{
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

int main()
{
	double *a, *b, *c;
	double *d_a, *d_b, *d_c;
	int size = N * sizeof(double);
	clock_t clock1, clock2;
	clock1 = clock();

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	a = new double[N] {1.};
	b = new double[N] {2.};
	c = new double[N} {3.};

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	add<<<N,1>>>(d_a, d_b, d_c);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);


	cout << "c[0] = " << a[0] << "  +  " << b[0] << "  =  " << c[0] << endl;


	
	clock2 = clock();
	cout << "clocks_per_sec : " << CLOCKS_PER_SEC << endl;
	cout << "time (ms) : " << 1000*(clock2 - clock1)/(double) CLOCKS_PER_SEC << endl;


	delete [] a; delete [] b; delete [] c;
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
}

