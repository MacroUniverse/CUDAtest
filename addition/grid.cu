
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

using namespace std;


__global__
void add(double *x, double *y, int N)
{
	int i, ind, stride;
	ind = blockIdx.x*blockDim.x + threadIdx.x;
	stride = gridDim.x * blockDim.x;
	for(i=ind; i<N; i+=stride) {
		y[i] += x[i];
	}
}

int main()
{
	double *d_x, *d_y, *x, *y, err{0.};
	int N = 1e6;
	int i, size = N*sizeof(double);
	hipMalloc((void **)&d_x, size);
	hipMalloc((void **)&d_y, size);
	x = new double[N];
	y = new double[N];
	for(i=0; i<N; ++i) {
		x[i] = 1.;
		y[i] = 2.;
	}
	hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

	add<<<20,512>>>(d_x, d_y, N);

	hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

	for (i=0; i<N; ++i) {
		err += (y[i]-3.)*(y[i]-3.);
	}
	cout << "err = " << err  << endl;

	delete [] x; delete [] y;
	hipFree(d_x); hipFree(d_y);

}
