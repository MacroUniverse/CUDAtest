
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
using namespace std;

__device__ double xmax = 3.14;
double xmax_;

__global__
void fun()
{
	printf(" xmax = %f\n", xmax);
}

int main()
{
	fun<<<1,1>>>();
	hipDeviceSynchronize();
	xmax_ = 5.;
	cout << "xmax_ = " << xmax_ << endl;
}
