
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
using namespace std;

__device__ double xmax = 3.14;
double xmax_;

__global__
void fun(int N)
{
	double *x = new double[N];
	double *y = new double[N];
	if (x) {
		x[N-1] = 3.1415926;
		y[N-1] = 2.7182818;
		//printf("x[N-1] = %f\n", x[N-1]);
	}
	else
		printf("heap overflow\n");
}

int main()
{
	int N = 1024;
	size_t heapsize;
	hipDeviceSetLimit(hipLimitMallocHeapSize, 41943040 + 96*1024*1024);
	hipDeviceGetLimit(&heapsize, hipLimitMallocHeapSize);
	cout << "heapsize = " << heapsize << endl;
	fun<<<1024*1024/256,256>>>(N);
	hipDeviceSynchronize();
	xmax_ = 5.;
}
