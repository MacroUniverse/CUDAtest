
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;


__global__
void add(double *x, double *y, int N)
{
	int i;
	for(i=0; i<N; ++i) {
		y[i] += x[i];
	}
}

int main()
{
	double *d_x, *d_y, *x, *y;
	int N = 1e6;
	int i, size = N*sizeof(double);
	hipMalloc((void **)&d_x, size);
	hipMalloc((void **)&d_y, size);
	x = new double[N];
	y = new double[N];
	for(i=0; i<N; ++i) {
		x[i] = 1.;
		y[i] = 2.;
	}
	hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

	add<<<1,1>>>(d_x, d_y, N);

	hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

	cout << y[0] << y[1] << y[2] << y[3] << endl;

	delete [] x; delete [] y;
	hipFree(d_x); hipFree(d_y);
}
