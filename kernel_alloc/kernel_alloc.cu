
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

using namespace std;

__global__
void fun(int N)
{
	double *a = new double[N];
	a[N-1] = 3.1415926;
	printf("a[] = %f\n", a[N-1]);
	delete[] a;
}

int main()
{
	int N = 10;
	fun<<<1,10>>>(N);
	hipDeviceSynchronize();

}
