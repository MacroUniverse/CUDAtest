
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

using namespace std;


__global__
void add(double *x, double *y, int N)
{
	int i, ind, stride;
	ind = blockIdx.x*blockDim.x + threadIdx.x;
	stride = gridDim.x * blockDim.x;
	for(i=ind; i<N; i+=stride) {
		++y[i];
		++x[i];
		++y[i];
		++x[i];
		++y[i];
		++x[i];
		++y[i];
		++x[i];
		++y[i];
		++y[i];
	}
}

int main()
{
	double *d_x, *d_y, *x, *y, err{0.};
	int N = 2e6;
	int i, size = N*sizeof(double);
	hipMalloc((void **)&d_x, size);
	hipMalloc((void **)&d_y, size);
	x = new double[N];
	y = new double[N];
	for(i=0; i<N; ++i) {
		x[i] = 3.1415;
		y[i] = 0.;
	}
	hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

	add<<<1280,64>>>(d_x, d_y, N);

	hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

	for (i=0; i<N; ++i) {
		err += (y[i]-6.)*(y[i]-6.);
	}
	cout << "err = " << err  << endl;

	delete [] x; delete [] y;
	hipFree(d_x); hipFree(d_y);

}
