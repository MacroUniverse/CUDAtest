// test the size of shared memory for each block


#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
using namespace std;

#define N 100

__const__ int NN = 1;

__global__
void fun(double *py)
{
	printf("NN = %d\n", NN);
	double a[NN];
	*py = 0.;
	for (int i=0; i<1000000; ++i)
		*py += 3.1415927;
}


int main()
{
	double *py, y;
	int N0 = 13;
	hipMemcpyToSymbol(HIP_SYMBOL(NN), &N0, sizeof(int));
	hipMalloc(&py, sizeof(double));
	fun<<<1,1>>>(py);
	hipMemcpy(&y, py, sizeof(double), hipMemcpyDeviceToHost);
	cout << "y = " << y << endl;
	hipDeviceSynchronize();
}
