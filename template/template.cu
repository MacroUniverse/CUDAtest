#include "hip/hip_runtime.h"
#ifdef _MSC_VER
#include "hip/hip_runtime.h"
#include ""
#endif

#include "nr3plus.h"
#include "matsave.h"
using std::cout; using std::endl; using std::string;
using std::ifstream; using std::to_string;

#define H2D hipMemcpyHostToDevice
#define D2H hipMemcpyDeviceToHost
#define cpySym hipMemcpyToSymbol

__global__
void kernel()
{
	printf("In kernel, block = %d, thread = %d\n", blockIdx.x, threadIdx.x);
}

int main()
{
	// cuda kernel call
	cout << "calling kernel..." << endl;
	kernel<<<2,2>>>();
	hipDeviceSynchronize();
	cout << "done calling kernel\n" << endl;

	// write data file
	cout << "writting data file..." << endl;
	MATFile *pfile = matOpen("nr.mat", "w");
	MatDoub A;
	A.resize(2, 3); A = 0.;
	A[0][0] = 1.; A[0][1] = 3.; A[0][2] = 5.; A[1][2] = 11;
	matsave(A, "A", pfile);
	matClose(pfile);
	cout << "done writing data file" << endl;
}
