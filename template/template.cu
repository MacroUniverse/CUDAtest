#include "hip/hip_runtime.h"
#include "nr3plus.h"
#include "matsave.h"
using std::cout; using std::endl; using std::string;
using std::ifstream; using std::to_string;

#define H2D hipMemcpyHostToDevice
#define D2H hipMemcpyDeviceToHost
#define cpySym hipMemcpyToSymbol

__global__
void kernel()
{
	printf("In kernel, block = %d, thread = %d\n", blockIdx.x, threadIdx.x);
}

int main()
{
	// cuda kernel call
	cout << "calling kernel..." << endl;
	kernel<<<2,2>>>();
	hipDeviceSynchronize();
	cout << "done calling kernel\n" << endl;

	// write data file
	cout << "writting data file..." << endl;
	MATFile *pfile = matOpen("nrMat.mat", "w");
	MatDoub A;
	A.assign(2, 3, 0.);
	A[0][0] = 1.; A[0][1] = 3.; A[0][2] = 5.; A[1][2] = 11;
	matsave(A, "A", pfile);
	matClose(pfile);
	cout << "done writing data file" << endl;
}
