// test the size of shared memory for each block


#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
using namespace std;

#define N 2500

__global__
void fun()
{
	__shared__ double x[N], y[N];

	for (int i=0; i<400; ++i)
	for (int j=0; j<N; ++j)
		y[j] += x[j];
}


int main()
{
	fun<<<1,1>>>();
	hipDeviceSynchronize();
}
