#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include "cuda_complex.hpp"

using namespace std;
typedef complex<double> Complex;

__global__
void add(Complex *x, Complex *y, int N)
{
	int i, ind, stride;
	ind = blockIdx.x*blockDim.x + threadIdx.x;
	stride = gridDim.x * blockDim.x;
	for(i=ind; i<N; i+=stride) {
		y[i] /= x[i];
	}
}

int main()
{
	Complex *d_x, *d_y, *x, *y;
	double err{0.};
	int N = 1e6;
	int i, size = N*sizeof(Complex);
	hipMalloc((void **)&d_x, size);
	hipMalloc((void **)&d_y, size);
	x = new Complex[N];
	y = new Complex[N];
	for(i=0; i<N; ++i) {
		x[i] = Complex(1.,1.);
		y[i] = Complex(2.,2.);
	}
	hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

	add<<<1,1>>>(d_x, d_y, N);

	hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

	for (i=0; i<N; ++i) {
		err += abs(y[i]-2.);
	}
	cout << "err = " << err << endl;

	delete [] x; delete [] y;
	hipFree(d_x); hipFree(d_y);

}
