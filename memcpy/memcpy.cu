
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__
void fun(double *a, long N)
{
	printf("inside fun()\n");
	printf("a[] = %f\n", a[N-1]);
}

int main()
{
	long i;
	long N = 2000000000;
	long size = N*sizeof(double);
	double *a = new double[N];
	double *a_d;

	cout << "hello" << endl;

	for(i=0; i<N; ++i)
		a[i] = 3.14159265358979323;

	cout << "assigned to a[]" << endl;

	hipMalloc(&a_d, size);

	cout << "cudaMalloc(); done" << endl;
	hipMemcpy(a_d, a, size, hipMemcpyHostToDevice);
	cout << "cudaMemcpy(); done" << endl;
	fun<<<1,1>>>(a_d, N);
	hipMemcpy(a, a_d, size, hipMemcpyDeviceToHost);
	delete[] a;

	hipFree(a_d);
}
